#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Type for points
typedef struct{
    float x;    // x coordinate
    float y;    // y coordinate
    int cluster; // cluster this point belongs to
} Point;

// Type for centroids
typedef struct{
    float x;    // x coordinate
    float y;    // y coordinate
    int nPoints; // number of points in this cluster
} Centroid;

// Global variables
int nPoints;   // Number of points
int nClusters; // Number of clusters/centroids

Point* points;       // Array containig all points
Centroid* centroids; // Array containing all centroids


// Reading command line arguments
void parse_args(int argc, char** argv){
    if(argc != 3){
        printf("Useage: kmeans nClusters nPoints\n");
        exit(-1);
    }
    nClusters = atoi(argv[1]);
    nPoints = atoi(argv[2]);
}


// Create random point
Point create_random_point(){
    Point p;
    p.x = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.y = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.cluster = rand() % nClusters;
    return p;
}


// Create random centroid
Centroid create_random_centroid(){
    Centroid p;
    p.x = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.y = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.nPoints = 0;
    return p;
}


// Initialize random data
// Points will be uniformly distributed
void init_data(){
    points = (Point*)malloc(sizeof(Point)*nPoints);
    for(int i = 0; i < nPoints; i++){
        points[i] = create_random_point();
        if(i < nClusters){
            points[i].cluster = i;
        }
    }

    centroids = (Centroid*)malloc(sizeof(Centroid)*nClusters);
    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }
}

// Initialize random data
// Points will be placed in circular clusters 
void init_clustered_data(){
    float diameter = 500.0/sqrt(nClusters);

    centroids = (Centroid*)malloc(sizeof(Centroid)*nClusters);
    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }

    points = (Point*)malloc(sizeof(Point)*nPoints);
    for(int i = 0; i < nPoints; i++){
        points[i] = create_random_point();
        if(i < nClusters){
            points[i].cluster = i;
        }
    }

    for(int i = 0; i < nPoints; i++){
        int c = points[i].cluster;
        points[i].x = centroids[c].x + ((float)rand() / (float)RAND_MAX) * diameter - (diameter/2);
        points[i].y = centroids[c].y + ((float)rand() / (float)RAND_MAX) * diameter - (diameter/2);
        points[i].cluster = rand() % nClusters;
    }

    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }
}


// Print all points and centroids to standard output
void print_data(){
    for(int i = 0; i < nPoints; i++){
        printf("%f\t%f\t%d\t\n", points[i].x, points[i].y, points[i].cluster);
    }
    printf("\n\n");
    for(int i = 0; i < nClusters; i++){
        printf("%f\t%f\t%d\t\n", centroids[i].x, centroids[i].y, i);
    }
}

// Print all points and centroids to a file
// File name will be based on input argument
// Can be used to print result after each iteration
void print_data_to_file(int i){
    char filename[15];
    sprintf(filename, "%04d.dat", i);
    FILE* f = fopen(filename, "w+");

    for(int i = 0; i < nPoints; i++){
        fprintf(f, "%f\t%f\t%d\t\n", points[i].x, points[i].y, points[i].cluster);
    }
    fprintf(f,"\n\n");
    for(int i = 0; i < nClusters; i++){
        fprintf(f,"%f\t%f\t%d\t\n", centroids[i].x, centroids[i].y, i);
    }

    fclose(f);
}


// Computing distance between point and centroid
float distance(Point a, Centroid b){
    float dx = a.x - b.x;
    float dy = a.y - b.y;

    return sqrt(dx*dx + dy*dy);
}


int main(int argc, char** argv){
    parse_args(argc, argv);

    // Create random data, either function can be used.
    //init_clustered_data();
    init_data();


    // Iterate until no points are updated
    int updated = 1;
    while(updated){
        updated = 0;

        // Reset centroid positions
        for(int i = 0; i < nClusters; i++){
            centroids[i].x = 0.0;
            centroids[i].y = 0.0;
            centroids[i].nPoints= 0;
        }


        // Compute new centroids positions
        for(int i = 0; i < nPoints; i++){
            int c = points[i].cluster;
            centroids[c].x += points[i].x;
            centroids[c].y += points[i].y;
            centroids[c].nPoints++;
        }

        for(int i = 0; i < nClusters; i++){
            // If a centroid lost all its points, we give it a random position
            // (to avoid dividing by 0)
            if(centroids[i].nPoints == 0){
                centroids[i] = create_random_centroid();
            }
            else{
                centroids[i].x /= centroids[i].nPoints;
                centroids[i].y /= centroids[i].nPoints;
            }
        }


        //Reassign points to closest centroid
        for(int i = 0; i < nPoints; i++){
            float bestDistance = DBL_MAX;
            int bestCluster = -1;

            for(int j = 0; j < nClusters; j++){
                float d = distance(points[i], centroids[j]);
                if(d < bestDistance){
                    bestDistance = d;
                    bestCluster = j;
                }
            }

            // If one point got reassigned to a new cluster, we have to do another iteration
            if(bestCluster != points[i].cluster){
                updated = 1;
            }
            points[i].cluster = bestCluster;
        }
    }

    print_data();
}
