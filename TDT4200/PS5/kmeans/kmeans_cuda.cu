#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

// Type for points
typedef struct{
    float x;    // x coordinate
    float y;    // y coordinate
    int cluster; // cluster this point belongs to
} Point;

// Type for centroids
typedef struct{
    float x;    // x coordinate
    float y;    // y coordinate
    int nPoints; // number of points in this cluster
} Centroid;

// Global variables
int nPoints;   // Number of points
int nClusters; // Number of clusters/centroids

Point* points;       // Array containig all points
Centroid* centroids; // Array containing all centroids


// Reading command line arguments
void parse_args(int argc, char** argv){
    if(argc != 3){
        printf("Useage: kmeans nClusters nPoints\n");
        exit(-1);
    }
    nClusters = atoi(argv[1]);
    nPoints = atoi(argv[2]);

    if (nPoints < 64 || nClusters < 1) {
        printf("nClusters must be greater than 0\nnPoints must be greater than or equal to 64\n");
        exit(1);
    }
}


// Create random point
Point create_random_point(){
    Point p;
    p.x = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.y = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.cluster = rand() % nClusters;
    return p;
}

// Create random centroid
Centroid create_random_centroid(){
    Centroid p;
    p.x = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.y = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.nPoints = 0;
    return p;
}

__host__ Centroid device_create_random_centroid(){
    Centroid c;
    c.x = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    c.y = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    c.nPoints = 0;
    return c;
}


// Initialize random data
// Points will be uniformly distributed
void init_data(){
    points = (Point*)malloc(sizeof(Point)*nPoints);
    for(int i = 0; i < nPoints; i++){
        points[i] = create_random_point();
        if(i < nClusters){
            points[i].cluster = i;
        }
    }

    centroids = (Centroid*)malloc(sizeof(Centroid)*nClusters);
    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }
}

// Initialize random data
// Points will be placed in circular clusters 
void init_clustered_data(){
    float diameter = 500.0/sqrt(nClusters);

    centroids = (Centroid*)malloc(sizeof(Centroid)*nClusters);
    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }

    points = (Point*)malloc(sizeof(Point)*nPoints);
    for(int i = 0; i < nPoints; i++){
        points[i] = create_random_point();
        if(i < nClusters){
            points[i].cluster = i;
        }
    }

    for(int i = 0; i < nPoints; i++){
        int c = points[i].cluster;
        points[i].x = centroids[c].x + ((float)rand() / (float)RAND_MAX) * diameter - (diameter/2);
        points[i].y = centroids[c].y + ((float)rand() / (float)RAND_MAX) * diameter - (diameter/2);
        points[i].cluster = rand() % nClusters;
    }

    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }
}


// Print all points and centroids to standard output
void print_data(){
    for(int i = 0; i < nPoints; i++){
        printf("%f\t%f\t%d\t\n", points[i].x, points[i].y, points[i].cluster);
    }
    printf("\n\n");
    for(int i = 0; i < nClusters; i++){
        printf("%f\t%f\t%d\t\n", centroids[i].x, centroids[i].y, i);
    }
}

// Print all points and centroids to a file
// File name will be based on input argument
// Can be used to print result after each iteration
void print_data_to_file(int i){
    char filename[15];
    sprintf(filename, "%04d.dat", i);
    FILE* f = fopen(filename, "w+");

    for(int i = 0; i < nPoints; i++){
        fprintf(f, "%f\t%f\t%d\t\n", points[i].x, points[i].y, points[i].cluster);
    }
    fprintf(f,"\n\n");
    for(int i = 0; i < nClusters; i++){
        fprintf(f,"%f\t%f\t%d\t\n", centroids[i].x, centroids[i].y, i);
    }

    fclose(f);
}


// Computing distance between point and centroid
float distance(Point a, Centroid b){
    float dx = a.x - b.x;
    float dy = a.y - b.y;

    return sqrt(dx*dx + dy*dy);
}

Point *input_points;
Centroid *input_centroids;
int *cuda_updated;
int *cuda_nClusters;
int *cuda_nPoints;

__global__ void device_reset_centroid_position(Centroid *input_centroids, Point *input_points, int *cuda_nPoints) {
    // extern __shared__ Point sdata[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int tid = threadIdx.x;
    // sdata[tid] = input_points[i];
    __syncthreads();

    input_centroids[i].x = 0.0;
    input_centroids[i].y = 0.0;
    input_centroids[i].nPoints = 0;

    // for (int s = 1; s < blockDim.x; s *= 2) {
    //     if (tid % (2 * s) == 0) {
    //         sdata[tid].x += sdata[tid + s].x;
    //         sdata[tid].y += sdata[tid + s].y;
    //         sdata[tid].nPoints += sdata[tid + s].nPoints;
    //     }
    //     __syncthreads();
    // }

    for(int j = 0; j < *cuda_nPoints; j++) {
        if (i == input_points[j].cluster) {
            input_centroids[i].x += input_points[j].x;
            input_centroids[i].y += input_points[j].y;
            input_centroids[i].nPoints += 1;
        }
    }
}

__global__ void device_reset_centroid_pos(Centroid *input_centroids) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    input_centroids[i].x = 0.0;
    input_centroids[i].y = 0.0;
    input_centroids[i].nPoints = 0;
}

__global__ void device_reassign_points(Point *d_points, Centroid *d_centroids, int *d_updated, int *d_nClusters, int *d_nPoints);

__global__ void device_compute_centroid_positions(Point *d_points, Centroid *d_centroids, int *d_nClusters, int *d_nPoints);

int main(int argc, char** argv) {
    srand(0);
    parse_args(argc, argv);

    // Create random data, either function can be used.
    //init_clustered_data();
    init_data();

    dim3 numBlocks, threadsPerBlock;
    numBlocks.x = nPoints / 64;
    threadsPerBlock.x = 64;

    // 1. Allocate buffers for the points and clusters
    hipMalloc((void**) &input_points, sizeof(Point) * nPoints);
    hipMalloc((void**) &input_centroids, sizeof(Centroid) * nClusters);
    hipMalloc((void**) &cuda_updated, sizeof(int));
    hipMalloc((void**) &cuda_nClusters, sizeof(int));
    hipMalloc((void**) &cuda_nPoints, sizeof(int));

    hipMemcpy(cuda_nClusters, &nClusters, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cuda_nPoints, &nPoints, sizeof(int), hipMemcpyHostToDevice);

    // Iterate until no points are updated
    int updated = 1;
    
    hipMemcpy(input_points, points, sizeof(Point) * nPoints, hipMemcpyHostToDevice);
    hipMemcpy(input_centroids, centroids, sizeof(Centroid) * nClusters, hipMemcpyHostToDevice);

    size_t smem_size = sizeof(Point) * 64 + sizeof(Centroid) * nClusters;

    while(updated) {
        updated = 0;

        if (nClusters < 0) {
            for (int i = 0; i < nClusters; i++) {
                centroids[i].x = 0.0;
                centroids[i].y = 0.0;
                centroids[i].nPoints= 0;
            }
            // Transfer the points and clusters to device
            hipMemcpy(input_points, points, sizeof(Point) * nPoints, hipMemcpyHostToDevice);
            hipMemcpy(input_centroids, centroids, sizeof(Centroid) * nClusters, hipMemcpyHostToDevice);
            // Reset centroid positions
            device_compute_centroid_positions<<<nClusters, threadsPerBlock, smem_size>>>(input_points, input_centroids, cuda_nClusters, cuda_nPoints);
            // Transfer data to host
            hipMemcpy(centroids, input_centroids, sizeof(Centroid) * nClusters, hipMemcpyDeviceToHost);
        } else {
            // Reset centroid positions
            for(int i = 0; i < nClusters; i++){
                centroids[i].x = 0.0;
                centroids[i].y = 0.0;
                centroids[i].nPoints= 0;
            }

            // Compute new centroids positions
            for(int i = 0; i < nPoints; i++){
                int c = points[i].cluster;
                centroids[c].x += points[i].x;
                centroids[c].y += points[i].y;
                centroids[c].nPoints++;
            }
        }

        // Because this function involes MATH rand(), it cannot be called from the kernel
        // By using the cuda_rand a random number could have been achieved, but it will
        // result in different result when running the host vs the cuda coda.
        for(int i = 0; i < nClusters; i++) {
            // If a centroid lost all its points, we give it a random position
            // (to avoid dividing by 0)
            if(centroids[i].nPoints == 0) {
                centroids[i] = create_random_centroid();
            }
            else {
                centroids[i].x /= centroids[i].nPoints;
                centroids[i].y /= centroids[i].nPoints;
            }
        }

        // Transfer the Centroids to the device, the Points is unchanged from the last time this function was called
        hipMemcpy(input_centroids, centroids, sizeof(Centroid) * nClusters, hipMemcpyHostToDevice);
        hipMemcpy(cuda_updated, &updated, sizeof(int), hipMemcpyHostToDevice);

        // Reassign points
        device_reassign_points<<<numBlocks, threadsPerBlock, smem_size>>>(input_points, input_centroids, cuda_updated, cuda_nClusters, cuda_nPoints);
        
        // Transfer the points back to the host, the Centroids remains unchanged
        hipMemcpy(points, input_points, sizeof(Point) * nPoints, hipMemcpyDeviceToHost);
        hipMemcpy(&updated, cuda_updated, sizeof(int), hipMemcpyDeviceToHost);

    }

    hipFree(input_points);
    hipFree(input_centroids);
    hipFree(cuda_updated);
    hipFree(cuda_nClusters);
    hipFree(cuda_nPoints);

    print_data();
}

__global__ void device_compute_centroid_positions(Point *d_points, Centroid *d_centroids, int *d_nClusters, int *d_nPoints) {
    extern __shared__ Point s[];
    Point *s_points = s;
    Centroid *s_centroids = (Centroid *) &s_points[64];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Init shared Points
    s_points[tid] = d_points[i];

    // Init shared Centroids
    int length = *d_nClusters / 64;

    // If number of clusters is small, let thread 0 do all the work, otherwise share between the 64 threads
    if (i == 0 && length < 1) {
        for (int j = 0; j < *d_nClusters; j++) {
            s_centroids[j] = (Centroid) {0.0, 0.0, 0};
        }
    } else if (length > 1) {
        int start = tid * length;
        int end = start + length;
        for (int j = start; j < end; j++) {
            s_centroids[j] = (Centroid) {0.0, 0.0, 0};
        }
    }
    __syncthreads();
    
    // if (i == 0) {
    //     printf("x:%f y:%f c:%d\n", d_centroids[0].x, d_centroids[0].y, d_centroids[0].nPoints);
    //     printf("x:%f y:%f c:%d\n", s_centroids[0].x, s_centroids[0].y, s_centroids[0].nPoints);
    // }

    int c = s_points[tid].cluster;
    atomicAdd(&s_centroids[c].x, s_points[tid].x);
    atomicAdd(&s_centroids[c].y, s_points[tid].y);
    atomicAdd(&s_centroids[c].nPoints, 1);
    __syncthreads();

    // // Reduce the Centroids
    // if (i < *d_nClusters) {
    //     atomicAdd(&d_centroids[i].x, s_centroids[i].x);
    //     atomicAdd(&d_centroids[i].y, s_centroids[i].y);
    //     atomicAdd(&d_centroids[i].nPoints, s_centroids[i].nPoints);
    // }
    if (tid == 0) {
        for(int x = 0; x < *d_nClusters; x++){
            atomicAdd(&d_centroids[x].x, s_centroids[x].x);
            atomicAdd(&d_centroids[x].y, s_centroids[x].y);
            atomicAdd(&d_centroids[x].nPoints, s_centroids[x].nPoints);
        }
    }
}

__global__ void device_reassign_points(Point *d_points, Centroid *d_centroids, int *d_updated, int *d_nClusters, int *d_nPoints) {
    // The shared memory consists of 64 points (1 for each thread per block), and 'd_nClusters' Centroids
    extern __shared__ Point s[];
    Point *s_points = s;
    Centroid *s_centroids = (Centroid *) &s_points[64];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Init shared Points
    s_points[tid] = d_points[i];

    // Init shared Centroids
    int length = *d_nClusters / 64;

    // If number of clusters is small, let thread 0 do all the work, otherwise share between the 64 threads
    if (tid == 0 && length < 1) {
        for (int b = 0; b < *d_nClusters; b++) {
            s_centroids[b] = d_centroids[b];
        }
    } else if (length > 1) {
        int start = tid * length;
        int end = start + length;
        for (int b = start; b < end; b++) {
            s_centroids[b] = d_centroids[b];
        }
    }
    __syncthreads();


    //Reassign points to closest centroid
    float bestDistance = DBL_MAX;
    int bestCluster = -1;

    for(int j = 0; j < *d_nClusters; j++) {
        Point a = s_points[tid];
        Centroid b = s_centroids[j];
        float dx = a.x - b.x;
        float dy = a.y - b.y;
        float d = sqrt(dx*dx + dy*dy);

        if(d < bestDistance) {
            bestDistance = d;
            bestCluster = j;
        }
    }

    // If one point got reassigned to a new cluster, we have to do another iteration
    if(bestCluster != s_points[tid].cluster) {
        *d_updated = 1;
    }

    // Update the points in the global memory
    d_points[i].cluster = bestCluster;
}
