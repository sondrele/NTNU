#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//#include "bmp.h"
extern "C" void write_bmp(unsigned char* data, int width, int height);
extern "C" unsigned char* read_bmp(char* filename);

//#include "host_blur.h"
extern "C" void host_blur(unsigned char* inputImage, unsigned char* outputImage, int size);

void print_properties() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    printf("Device count: %d\n", deviceCount);

    hipDeviceProp_t p;
    hipSetDevice(0);
    hipGetDeviceProperties (&p, 0);

    printf("Compute capability: %d.%d\n", p.major, p.minor);
    printf("Name: %s\n" , p.name);
    printf("\n\n");
}

__global__ void device_blur(unsigned char *input_img, unsigned char *output_img) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + y * 512; 

    output_img[index] = 0;
    for(int k = -1; k < 2; k++) {
        for(int l = -1; l < 2; l++) {
            // TOOD: Add 1px border to input_img and fix index
            output_img[index] += (input_img[index + k + l] / 9.0);
        }
    }
}

int main(int argc,char **argv) {
    // Prints some device properties, also to make sure the GPU works etc.
    print_properties();

    //Currently we do the bluring on the CPU
    unsigned char *A = read_bmp("peppers.bmp");
    unsigned char *B = (unsigned char *) malloc(sizeof(unsigned char) * 512 * 512);

    dim3 numBlocks, threadsPerBlock;

    numBlocks.x = 64; numBlocks.y = 64; // 4096 blocks
    threadsPerBlock.x = 8; threadsPerBlock.y = 8; // 64 threads per block

    // 1. Allocate buffers for the input image and the output image
    unsigned char *input_img;
    hipMalloc((void**) &input_img, sizeof(unsigned char) * 512 * 512);
    printf("1: %s \n", hipGetErrorString(hipGetLastError()));

    unsigned char *output_img;
    hipMalloc((void**) &output_img, sizeof(unsigned char) * 512 * 512);
    printf("2: %s \n", hipGetErrorString(hipGetLastError()));

    // 2. Transfer the input image from the host to the device
    hipMemcpy(input_img, A, sizeof(unsigned char) * 512 * 512, hipMemcpyHostToDevice);
    printf("3: %s \n", hipGetErrorString(hipGetLastError()));

    // 3. Launch the kernel which does the bluring
    device_blur<<<numBlocks, threadsPerBlock>>>(input_img, output_img);
    printf("4: %s \n", hipGetErrorString(hipGetLastError()));

    // 4. Transfer the result back to the host.
    hipMemcpy(B, output_img, sizeof(unsigned char) * 512 * 512, hipMemcpyDeviceToHost);
    printf("5: %s \n", hipGetErrorString(hipGetLastError()));

    write_bmp(B, 512, 512);

    free(A);
    free(B);

    return 0;
}
